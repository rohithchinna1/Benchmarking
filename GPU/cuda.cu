#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#define N 1024

__global__ void add( int *a, int *b, int *c ) {
 if(threadIdx.x<N)
c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__global__ void add1( float  *a, float  *b, float  *c ) {
 if(threadIdx.x<N)
c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

long getMicroSeconds();
double noofOperations,totalTime,IOPS,GFLOPS,FLOPS,GIOPS,totalTime1,IFLOPS;


int main( void ) {
float a,b,c;
int a1[N], b1[N], c1[N];
float *dev_a,*dev_b,*dev_c;
int *dev_a1, *dev_b1, *dev_c1;
double  time=0;
int size=sizeof(int);
int fsize=sizeof(float);
double start,stop,end,start1,end1,time1;
int i;
int threadCount=0;
hipMalloc( (void**)&dev_a, N * sizeof(float) );
hipMalloc( (void**)&dev_b, N * sizeof(float) );
hipMalloc( (void**)&dev_c, N * sizeof(float) );

a=2.3;
b=4.2;
int istart = getMicroSeconds();
hipMemcpy(dev_a, &a,fsize, hipMemcpyHostToDevice);
stop=getMicroSeconds();
time=fsize/(stop-start);
printf("\n Read bandwidth for 1B%f\t\n",time);
hipMemcpy(dev_b, &b,fsize, hipMemcpyHostToDevice);
start = getMicroSeconds();
add1<<<1,1>>>(dev_a, dev_b, dev_c);
threadCount=1;
noofOperations=1;
stop=getMicroSeconds();
totalTime=fsize/(stop-start);
printf("\n Write Bandwidth of 1B %f\n",totalTime);
FLOPS=noofOperations/totalTime;
printf("\n FLOPS\t%f",FLOPS);
printf("\tThreadCount\t\%d\n",threadCount);

GFLOPS=FLOPS/(pow(10,9));
printf("time taken in gflops: %f\n",GFLOPS);

hipMemcpy( c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost );
printf("c %f \n",c);

hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

//************1 KB byte************************      
// allocate the memory on the GPU
hipMalloc( (void**)&dev_a1, N * sizeof(int) ) ;
hipMalloc( (void**)&dev_b1, N * sizeof(int) ) ;
hipMalloc( (void**)&dev_c1, N * sizeof(int) ) ;

 // fill the arrays 'a' and 'b' on the CPU
for (i=0; i<N; i++)
{
a1[i] = -i;
b1[i] = i * i;
}

time=0;
start=getMicroSeconds();

hipMemcpy( dev_a1, a1, N * sizeof(int), hipMemcpyHostToDevice );
hipMemcpy( dev_b1, b1, N * sizeof(int), hipMemcpyHostToDevice );
  end=getMicroSeconds();
  time=(2*size)/(end-start);
  printf("\n Read bandwidth for 1KB%f\t\n",time);
  start=getMicroSeconds();
       add<<<1,N>>>( dev_a1, dev_b1, dev_c1 );
       stop=getMicroSeconds();
   start1=getMicroSeconds();
  hipMemcpy( c1, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost );
  end1=getMicroSeconds();
  time1=(end1-start1);
  totalTime1=(2*size)/time1;
  printf("\n Write Bandwidthof 1KB %f\n",totalTime1);


  time=(stop-start);
  threadCount=N;
  noofOperations=1;
  totalTime=(2*size)/time;
  IOPS=noofOperations/totalTime;
  printf("\n IOPS\t%f\t",IOPS);
  printf("\ThreadCount\t\%d\n",threadCount);
  IFLOPS=IOPS/(pow(10,9));
  printf("time taken in Iflops: %f\n",IFLOPS);

  start1=getMicroSeconds(); 
  hipMemcpy( c1, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost );
  end1=getMicroSeconds();
  time1=(end1-start1);

totalTime1=(2*size)/time1;
  printf("\n Write Bandwidth %f\n",totalTime1);


       // display the results
  for (int i=0; i<N; i++) {
   printf( "%d + %d = %d\n", a1[i], b1[i], c1[i] );
   }
       // free the memory allocated on the GPU
       hipFree( dev_a1 );
       hipFree( dev_b1 );
       hipFree( dev_c1 );
return 0;
}


 long getMicroSeconds(){
        struct timeval tv;
        gettimeofday(&tv, NULL);
        return tv.tv_sec * 1000000 + tv.tv_usec;
}

